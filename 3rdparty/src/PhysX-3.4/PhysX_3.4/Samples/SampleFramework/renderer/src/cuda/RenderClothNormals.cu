#include "hip/hip_runtime.h"
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Copyright (c) 2008-2019 NVIDIA Corporation. All rights reserved.

#include "PxPhysics.h"
#include "PxVec4.h"
#include "PxVec3.h"
#include "PxVec2.h"

#include "cloth/PxClothTypes.h"

namespace physx
{

// interleaved format must match that used by RendererClothShape
struct Vertex
{
	PxVec3 position;
	PxVec3 normal;
};

namespace
{
	__device__ inline void PxAtomicFloatAdd(float* dest, float x)
	{
#if __CUDA_ARCH__ >= 200
		atomicAdd(dest, x);
#else
		union bits { float f; unsigned int i; };
		bits oldVal, newVal;

		do
		{
			// emulate atomic float add on 1.1 arch
			oldVal.f = *dest;
			newVal.f = oldVal.f + x;
		}
		while (atomicCAS((unsigned int*)dest, oldVal.i, newVal.i) != oldVal.i);
#endif
	}


	__device__ void PxAtomicVec3Add(PxVec3& dest, PxVec3 inc)
	{
		PxAtomicFloatAdd(&dest.x, inc.x);
		PxAtomicFloatAdd(&dest.y, inc.y);
		PxAtomicFloatAdd(&dest.z, inc.z);
	}
}

extern "C" __global__ void computeSmoothNormals(
	const PxClothParticle* particles,
	const PxU16* indices,
	Vertex* vertices,
	PxU32 numTris,
	PxU32 numParticles)
{
	// zero old normals
	for (PxU32 i=threadIdx.x; i < numParticles; i += blockDim.x)
		vertices[i].normal = PxVec3(0.0f);

	__syncthreads();

	for (PxU32 i=threadIdx.x; i < numTris; i += blockDim.x)
	{
		PxU16 a = indices[i*3];
		PxU16 b = indices[i*3+1];
		PxU16 c = indices[i*3+2];

		// calculate face normal
		PxVec3 e1 = particles[b].pos-particles[a].pos;
		PxVec3 e2 = particles[c].pos-particles[a].pos;
		PxVec3 n = e2.cross(e1);

		PxAtomicVec3Add(vertices[a].normal, n);
		PxAtomicVec3Add(vertices[b].normal, n);
		PxAtomicVec3Add(vertices[c].normal, n);
	}

	__syncthreads();

	// update vertex buffer
	for (PxU32 i=threadIdx.x; i < numParticles; i += blockDim.x)
	{
		vertices[i].position = particles[i].pos;
		vertices[i].normal = vertices[i].normal.getNormalized();
	}	
}

}