#include "hip/hip_runtime.h"
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Copyright (c) 2008-2019 NVIDIA Corporation. All rights reserved.

#include "PxPhysics.h"
#include "PxVec4.h"
#include "PxVec3.h"
#include "PxVec2.h"
#include "PxMat33.h"
#include "PxStrideIterator.h"

namespace physx
{

template <typename T>
__device__ T* ptrOffset(T* p, PxU32 byteOffset)
{
	return (T*)((unsigned char*)(p) + byteOffset);
}

#if __CUDA_ARCH__ < 200
__device__ PxU32 gOffset;
#else
__device__ __shared__ PxU32 gOffset;
#endif


// copies orientations and positions to the destination vertex
// buffer based on the validityBitmap state
extern "C" __global__ void updateInstancedVB(
	PxVec3* destPositions,
	PxVec3* destRotation0,
	PxVec3* destRotation1,
	PxVec3* destRotation2,
	PxU32 destStride,
	const PxVec4* srcPositions,
	const PxMat33* srcRotations,
	const PxU32* validParticleBitmap,
	PxU32 validParticleRange)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));
			
			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				const PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*destStride;

				*ptrOffset(destRotation0, offset) = srcRotations[index].column0;
				*ptrOffset(destRotation1, offset) = srcRotations[index].column1;
				*ptrOffset(destRotation2, offset) = srcRotations[index].column2;

				PxVec3* p = ptrOffset(destPositions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				++destIndex;
			}
		}
	}
}


// copies positions and alpha to the destination vertex buffer based on 
// validity bitmap and particle life times
extern "C" __global__ void updateBillboardVB(
	PxVec3* destPositions,
	PxU8* destAlphas,
	PxU32 destStride,
	PxF32 fadingPeriod,
	const PxVec4* srcPositions, 
	const PxReal* srcLifetimes,
	const PxU32* validParticleBitmap,
	PxU32 validParticleRange)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));

			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*destStride;

				// copy position
				PxVec3* p = ptrOffset(destPositions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				// update alpha
				if (srcLifetimes)
				{
					PxU8 lifetime = 0;
					if(srcLifetimes[index] >= fadingPeriod)
						lifetime = 255;
					else
					{
						if(srcLifetimes[index] <= 0.0f)
							lifetime = 0; 
						else
							lifetime = static_cast<PxU8>(srcLifetimes[index] * 255 / fadingPeriod);
					}

					destAlphas[destIndex*4] = lifetime;
				}

				++destIndex;
			}
		}
	}
}

}